
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__global__ void count(int *key,int *bucket){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  atomicAdd(&bucket[key[i]+1], 1);
}

__global__ void scan(int *bucket,int *tmp,int n){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  for(int j=1; j<n; j<<=1) {
    tmp[i+1] = bucket[i+1];
    __syncthreads();
    if(i>=j) bucket[i+1] += tmp[i-j+1];
    __syncthreads();
  }
}

__global__ void put(int *key,int *bucket){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  for(int j = bucket[i]; j <bucket[i+1];j++){
    key[j] = i;
  }
}

int main() {
  int n = 50;
  int range = 5;
  int *key, *bucket, *tmp;
  hipMallocManaged(&key, n*sizeof(int));
  hipMallocManaged(&bucket, (range+1)*sizeof(int));
  hipMallocManaged(&tmp, (range+1)*sizeof(int));
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");

  //std::vector<int> bucket(range); 
  for (int i=0; i<range+1; i++) {
    bucket[i] = 0;
    tmp[i] = 0;
  }

  count<<<1,n>>>(key,bucket);
  hipDeviceSynchronize();
  scan<<<1,range>>>(bucket,tmp,range);
  hipDeviceSynchronize();
  put<<<1,range>>>(key,bucket);
  hipDeviceSynchronize();

  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");
}
